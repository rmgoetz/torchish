#include "hip/hip_runtime.h"

#include "raycast.hpp"
#include "../vector_f.cuh"

__global__ void raycast_kernel(
    torch::PackedTensorAccessor32<float, 3> origins,
    torch::PackedTensorAccessor32<float, 3> directions,
    torch::PackedTensorAccessor32<float, 2> vertices,
    torch::PackedTensorAccessor32<int64_t, 2> faces,
    torch::PackedTensorAccessor32<int64_t, 1> faces_per_batch,
    torch::PackedTensorAccessor32<int64_t, 1> face_index_start,
    torch::PackedTensorAccessor32<float, 2> distances,
    torch::PackedTensorAccessor32<float, 3> normals,
    const int B,
    const int R,
    const int F,
    const int threads_per_block)
{
    // The index for this thread within the block
    const uint32_t thread_index = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);

    // The first block dimension corresponds to the batch index this thread works for
    const uint32_t batch_index = blockIdx.x;

    // The second block dimension corresponds to a set of threads_per_block rays
    const uint32_t ray_index = thread_index + threads_per_block * blockIdx.y;

    // Likely not much of a performance impact, but leverage a small shared memory
    // SHARED[0] -> first face index for this batch
    // SHARED[1] -> number of faces in this batch
    __shared__ int64_t SHARED[2];
    if (thread_index == 0)
    {
        SHARED[thread_index] = face_index_start[batch_index];
    }
    else if (thread_index == 1)
    {
        SHARED[thread_index] = faces_per_batch[batch_index];
    }
    __syncthreads();

    // Do nothing for the thread outside of the ray bounds
    if (ray_index >= R)
    {
        return;
    }

    //
    float *origin_ptr = &origins[batch_index][ray_index][0];
    float *direction_ptr = &directions[batch_index][ray_index][0];
    cudVec3D_f origin = cudVec3D_f(origin_ptr);
    cudVec3D_f direction = cudVec3D_f(direction_ptr);

    //
    const int64_t face_start = SHARED[0];
    const int64_t face_stop = face_start + SHARED[1];

    //
    float infinity = std::numeric_limits<float>::infinity();
    float min_distance = infinity;

    //
    float eps1 = 1e-7;
    float eps2 = 1e-4;

    //
    float closest_face_normal_x;
    float closest_face_normal_y;
    float closest_face_normal_z;

    //
    cudVec3D_f edge_0_1;
    cudVec3D_f edge_0_2;
    cudVec3D_f face_normal;
    cudVec3D_f rayd_cross_shifto;
    float n_dot_rayd;
    float t;
    float scale;
    float alpha;
    float beta;
    float gamma;

    //
    for (int face_index = face_start; face_index < face_stop; face_index++)
    {
        //
        float *vertex_0_ptr = &vertices[faces[face_index][0]][0];
        float *vertex_1_ptr = &vertices[faces[face_index][1]][0];
        float *vertex_2_ptr = &vertices[faces[face_index][2]][0];
        cudVec3D_f vertex_0 = cudVec3D_f(vertex_0_ptr);
        cudVec3D_f vertex_1 = cudVec3D_f(vertex_1_ptr);
        cudVec3D_f vertex_2 = cudVec3D_f(vertex_2_ptr);

        //
        edge_0_1 = vertex_1 - vertex_0;
        edge_0_2 = vertex_2 - vertex_0;
        face_normal = edge_0_1.cross(edge_0_2);

        //
        n_dot_rayd = face_normal.dot(direction);
        if (fabs(n_dot_rayd) < eps1)
        {
            continue;
        }

        //
        t = (vertex_0 - origin).dot(face_normal) / n_dot_rayd;

        //
        if (t < -eps1)
        {
            continue;
        }

        //
        rayd_cross_shifto = direction.cross(origin - vertex_0);
        scale = direction.dot(face_normal);
        beta = edge_0_2.dot(rayd_cross_shifto) / scale;
        gamma = -edge_0_1.dot(rayd_cross_shifto) / scale;
        alpha = 1 - beta - gamma;

        //
        bool in_face = (alpha > -eps2) && (alpha < 1 + eps2) && (beta > -eps2) && (beta < 1 + eps2) && (gamma > -eps2) && (gamma < 1 + eps2);

        //
        if (!in_face)
        {
            continue;
        }

        //
        if (t < min_distance)
        {
            //
            min_distance = t;

            //
            face_normal = face_normal.normalize();

            //
            const int sign = (face_normal.dot(direction) > 0) ? -1 : 1;
            closest_face_normal_x = face_normal[0] * sign;
            closest_face_normal_y = face_normal[1] * sign;
            closest_face_normal_z = face_normal[2] * sign;
        }
    }

    //
    if (min_distance < infinity)
    {
        distances[batch_index][ray_index] = min_distance;
        normals[batch_index][ray_index][0] = closest_face_normal_x;
        normals[batch_index][ray_index][1] = closest_face_normal_y;
        normals[batch_index][ray_index][2] = closest_face_normal_z;
    }
}

std::vector<torch::Tensor> raycast_CUDA(
    torch::Tensor origins,      // [B, R, 3 (x, y, z)]
    torch::Tensor directions,   // [B, R, 3 (x, y, z)]
    torch::Tensor vertices,     // [V, 3 (x, y, z)]
    torch::Tensor faces,        // [F, 3 (v0, v1, v2)]
    torch::Tensor vertex_batch) // [V] consecutive and sorted
{
    using namespace torch::indexing;

    // The number of batches of rays
    const uint32_t B = origins.size(0);

    // The number of rays
    const uint32_t R = origins.size(1);

    // The number of total triangulated mesh facets across all batches
    const uint32_t F = faces.size(0);

    // The tensor corresponding each facet to a batch index
    torch::Tensor face_batch = vertex_batch.index({faces.index({Slice(), 0})}).contiguous(); // [F]

    // The number of faces in each batch
    std::tuple<at::Tensor, at::Tensor, at::Tensor> unique_result = at::_unique2(face_batch, true, false, true);
    torch::Tensor faces_per_batch = std::get<2>(unique_result); // [B]

    // The index of the first face for each batch index
    torch::Tensor face_index_start = torch::cat({torch::zeros({1}, faces_per_batch.options()),
                                                 torch::cumsum(faces_per_batch, 0).index({Slice{0, B - 1}})},
                                                0); // [B]

    // Instantiate the distances tensor and
    float inf = std::numeric_limits<float>::infinity();
    torch::Tensor distances = torch::full({B, R}, inf, origins.options());  // [B, R]
    torch::Tensor normals = torch::full({B, R, 3}, inf, origins.options()); // [B, R, 3 (x, y, z)]

    //
    dim3 threads(256, 1, 1);
    const int threads_per_block = threads.x * threads.y * threads.z;
    const int R_blocks = R / threads_per_block + ((R % threads_per_block != 0) ? 1 : 0);
    dim3 blocks(B, R_blocks, 1);

    // Launch the kernel
    raycast_kernel<<<blocks, threads>>>(
        origins.packed_accessor32<float, 3>(),
        directions.packed_accessor32<float, 3>(),
        vertices.packed_accessor32<float, 2>(),
        faces.packed_accessor32<int64_t, 2>(),
        faces_per_batch.packed_accessor32<int64_t, 1>(),
        face_index_start.packed_accessor32<int64_t, 1>(),
        distances.packed_accessor32<float, 2>(),
        normals.packed_accessor32<float, 3>(),
        B,
        R,
        F,
        threads_per_block);

    return {distances, normals}; // [B, R], [B, R, 3 (x, y, z)]
}
